#include "hip/hip_runtime.h"
#include "left_right_consistency.h"

__global__ void ChooseRightDisparity(uint8_t *right_disparity, const uint16_t *smoothed_cost, const uint32_t rows, const uint32_t cols)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols || y >= rows)
        return;

    int min_cost_disparity = 0;
    uint16_t min_cost = smoothed_cost[(y * cols + x) * MAX_DISPARITY + min_cost_disparity];

    for (int d = 1; d < MAX_DISPARITY; d++)
    {
        if (x + d >= cols)
            break;
        uint16_t tmp_cost = smoothed_cost[(y * cols + (x + d)) * MAX_DISPARITY + d];
        if (tmp_cost < min_cost)
        {
            min_cost = tmp_cost;
            min_cost_disparity = d;
        }
    }

    right_disparity[y * cols + x] = min_cost_disparity;
}

__global__ void LeftRightConsistencyCheck(uint8_t *disparity, const uint8_t *disparity_right, uint32_t rows, uint32_t cols)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols || y >= rows)
        return;

    const int x_right = x - disparity[y * cols + x];

    if (x_right < 0)
    {
        disparity[y * cols + x] = 255;
        return;
    }

    int diff = disparity[y * cols + x] - disparity_right[y * cols + x_right];
    diff = diff < 0 ? diff * -1 : diff;
    if (diff > 1)
    {
        disparity[y * cols + x] = 255;
    }
}
